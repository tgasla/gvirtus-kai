#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <vector>
#include <cmath>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)
#define CUFFT_CHECK(err) ASSERT_EQ((err), HIPFFT_SUCCESS)

TEST(cuFFT, cufftCreateDestroy) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));

    // Check if the plan was created successfully
    ASSERT_NE(plan, 0);

    // Clean up
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, Plan1D) {
    const int N = 8;
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

    // Check if the plan was created successfully
    ASSERT_NE(plan, 0);

    // Clean up
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, Plan2D) {
    const int NX = 4, NY = 4;
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C));

    // Check if the plan was created successfully
    ASSERT_NE(plan, 0);

    // Clean up
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, Plan3D) {
    const int NX = 4, NY = 4, NZ = 4;
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C));
    ASSERT_NE(plan, 0);
    CUFFT_CHECK(hipfftDestroy(plan));
}

// TEST(cuFFT, PlanMany) {
//     const int rank = 1;
//     int n[] = {8};
//     const int howmany = 2;
//     hipfftHandle plan;
//     CUFFT_CHECK(hipfftPlanMany(&plan, rank, n,
//                               nullptr, 1, 0,  // inembed, istride, idist
//                               nullptr, 1, 0,  // onembed, ostride, odist
//                               HIPFFT_C2C, howmany));
//     ASSERT_NE(plan, 0);
//     CUFFT_CHECK(hipfftDestroy(plan));
// }

TEST(cuFFT, Estimate1D) {
    size_t workSize = 0;
    CUFFT_CHECK(hipfftEstimate1d(8, HIPFFT_C2C, 1, &workSize));
    ASSERT_GT(workSize, 0u);
}

TEST(cuFFT, Estimate2D) {
    size_t workSize = 0;
    CUFFT_CHECK(hipfftEstimate2d(4, 4, HIPFFT_C2C, &workSize));
    ASSERT_GT(workSize, 0u);
}

TEST(cuFFT, Estimate3D) {
    size_t workSize = 0;
    CUFFT_CHECK(hipfftEstimate3d(4, 4, 4, HIPFFT_C2C, &workSize));
    ASSERT_GT(workSize, 0u);
}

TEST(cuFFT, EstimateMany) {
    int n[] = {8};
    size_t workSize = 0;
    CUFFT_CHECK(hipfftEstimateMany(1, n,
                                  nullptr, 1, 0,
                                  nullptr, 1, 0,
                                  HIPFFT_C2C, 2,
                                  &workSize));
    ASSERT_GT(workSize, 0u);
}

TEST(cuFFT, Plan1DAndExecC2C) {
    const int N = 8;
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

    std::vector<hipfftComplex> input(N), output(N);

    // Fill input with known values
    for (int i = 0; i < N; ++i) {
        input[i].x = static_cast<float>(i);
        input[i].y = 0.0f;
    }

    hipfftComplex *d_input;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(hipfftComplex) * N));
    CUDA_CHECK(hipMemcpy(d_input, input.data(), sizeof(hipfftComplex) * N, hipMemcpyHostToDevice));

    CUFFT_CHECK(hipfftExecC2C(plan, d_input, d_input, HIPFFT_FORWARD));
    CUDA_CHECK(hipMemcpy(output.data(), d_input, sizeof(hipfftComplex) * N, hipMemcpyDeviceToHost));

    // Optional: Check at least that output is not all zero
    float totalEnergy = 0.0f;
    for (auto &v : output) {
        totalEnergy += v.x * v.x + v.y * v.y;
    }
    ASSERT_GT(totalEnergy, 0.0f);

    CUFFT_CHECK(hipfftDestroy(plan));
    CUDA_CHECK(hipFree(d_input));
}

TEST(cuFFT, CreateMakePlan1dAndExecC2C) {
    const int N = 8;  // FFT size

    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));

    size_t workSize = 0;
    CUFFT_CHECK(hipfftMakePlan1d(plan, N, HIPFFT_C2C, 1, &workSize));

    hipfftComplex *d_in, *d_out;
    CUDA_CHECK(hipMalloc(&d_in, sizeof(hipfftComplex) * N));
    CUDA_CHECK(hipMalloc(&d_out, sizeof(hipfftComplex) * N));

    // Initialize input on host
    hipfftComplex h_in[N];
    for (int i = 0; i < N; ++i) {
        h_in[i].x = static_cast<float>(i);
        h_in[i].y = 0.0f;
    }

    CUDA_CHECK(hipMemcpy(d_in, h_in, sizeof(h_in), hipMemcpyHostToDevice));

    // Execute FFT
    CUFFT_CHECK(hipfftExecC2C(plan, d_in, d_out, HIPFFT_FORWARD));

    // Copy result back
    hipfftComplex h_out[N];
    CUDA_CHECK(hipMemcpy(h_out, d_out, sizeof(h_out), hipMemcpyDeviceToHost));

    // Basic check: output should not all be zero
    bool all_zero = true;
    for (int i = 0; i < N; ++i) {
        if (h_out[i].x != 0 || h_out[i].y != 0) {
            all_zero = false;
            break;
        }
    }
    ASSERT_FALSE(all_zero) << "All FFT output values are zero — transform likely failed.";

    // Cleanup
    CUFFT_CHECK(hipfftDestroy(plan));
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
}

TEST(cuFFT, MakePlan2D) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    size_t workSize = 0;
    CUFFT_CHECK(hipfftMakePlan2d(plan, 4, 4, HIPFFT_C2C, &workSize));
    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, MakePlan3D) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    size_t workSize = 0;
    CUFFT_CHECK(hipfftMakePlan3d(plan, 4, 4, 4, HIPFFT_C2C, &workSize));
    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, MakePlanMany) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));

    int n[] = {8};
    size_t workSize = 0;

    CUFFT_CHECK(hipfftMakePlanMany(plan, 1, n,
                                  nullptr, 1, 0,
                                  nullptr, 1, 0,
                                  HIPFFT_C2C, 2, &workSize));

    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}


#if CUDART_VERSION >= 7000
TEST(cuFFT, MakePlanMany64) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));

    long long int n[] = {8};
    long long int inembed[] = {8};
    long long int onembed[] = {8};
    long long int istride = 1, ostride = 1;
    long long int idist = 8, odist = 8;
    long long int howmany = 2;
    size_t workSize = 0;

    CUFFT_CHECK(hipfftMakePlanMany64(plan, 1, n,
                                    inembed, istride, idist,
                                    onembed, ostride, odist,
                                    HIPFFT_C2C, howmany, &workSize));

    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}
#endif

TEST(cuFFT, Plan2DAndExecC2C) {
    const int NX = 4, NY = 4;
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C));

    std::vector<hipfftComplex> input(NX * NY), output(NX * NY);

    for (int i = 0; i < NX * NY; ++i) {
        input[i].x = static_cast<float>(i);
        input[i].y = 0.0f;
    }

    hipfftComplex *d_input;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(hipfftComplex) * NX * NY));
    CUDA_CHECK(hipMemcpy(d_input, input.data(), sizeof(hipfftComplex) * NX * NY, hipMemcpyHostToDevice));

    CUFFT_CHECK(hipfftExecC2C(plan, d_input, d_input, HIPFFT_FORWARD));
    CUDA_CHECK(hipMemcpy(output.data(), d_input, sizeof(hipfftComplex) * NX * NY, hipMemcpyDeviceToHost));

    // Check for non-zero output
    float energy = 0;
    for (auto &v : output) {
        energy += v.x * v.x + v.y * v.y;
    }
    ASSERT_GT(energy, 0.0f);

    CUFFT_CHECK(hipfftDestroy(plan));
    CUDA_CHECK(hipFree(d_input));
}

TEST(cuFFT, ExecR2CAndC2R) {
    const int N = 16;
    hipfftHandle plan_fwd, plan_inv;

    CUFFT_CHECK(hipfftPlan1d(&plan_fwd, N, HIPFFT_R2C, 1));
    CUFFT_CHECK(hipfftPlan1d(&plan_inv, N, HIPFFT_C2R, 1));

    std::vector<float> input(N);
    std::vector<float> output(N);
    std::vector<hipfftComplex> spectrum(N / 2 + 1);

    for (int i = 0; i < N; ++i) input[i] = sin(2 * M_PI * i / N); // A sine wave

    float *d_input;
    hipfftComplex *d_spectrum;
    float *d_output;

    CUDA_CHECK(hipMalloc(&d_input, sizeof(float) * N));
    CUDA_CHECK(hipMalloc(&d_spectrum, sizeof(hipfftComplex) * (N / 2 + 1)));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(float) * N));

    CUDA_CHECK(hipMemcpy(d_input, input.data(), sizeof(float) * N, hipMemcpyHostToDevice));

    CUFFT_CHECK(hipfftExecR2C(plan_fwd, d_input, d_spectrum));
    CUFFT_CHECK(hipfftExecC2R(plan_inv, d_spectrum, d_output));

    CUDA_CHECK(hipMemcpy(output.data(), d_output, sizeof(float) * N, hipMemcpyDeviceToHost));

    // Normalize inverse FFT output
    for (int i = 0; i < N; ++i)
        output[i] /= N;

    for (int i = 0; i < N; ++i)
        ASSERT_NEAR(input[i], output[i], 1e-3f);

    CUFFT_CHECK(hipfftDestroy(plan_fwd));
    CUFFT_CHECK(hipfftDestroy(plan_inv));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_spectrum));
    CUDA_CHECK(hipFree(d_output));
}

TEST(cuFFT, GetSize1D) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    size_t workSize = 0;
    CUFFT_CHECK(hipfftGetSize1d(plan, 8, HIPFFT_C2C, 1, &workSize));
    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, GetSize2D) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    size_t workSize = 0;
    CUFFT_CHECK(hipfftGetSize2d(plan, 4, 4, HIPFFT_C2C, &workSize));
    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, GetSize3D) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    size_t workSize = 0;
    CUFFT_CHECK(hipfftGetSize3d(plan, 4, 4, 4, HIPFFT_C2C, &workSize));
    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, GetSizeMany) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    int n[] = {8};
    size_t workSize = 0;
    CUFFT_CHECK(hipfftGetSizeMany(plan, 1, n,
                                 nullptr, 1, 0,
                                 nullptr, 1, 0,
                                 HIPFFT_C2C, 2, &workSize));
    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}

#if CUDART_VERSION >= 7000
TEST(cuFFT, GetSizeMany64) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    long long int n[] = {8};
    long long int inembed[] = {8};
    long long int onembed[] = {8};
    size_t workSize = 0;
    CUFFT_CHECK(hipfftGetSizeMany64(plan, 1, n,
                                   inembed, 1, 8,
                                   onembed, 1, 8,
                                   HIPFFT_C2C, 2, &workSize));
    ASSERT_GT(workSize, 0u);
    CUFFT_CHECK(hipfftDestroy(plan));
}
#endif

TEST(cuFFT, GetSize) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    int n[] = {8};
    size_t workSize = 0;
    CUFFT_CHECK(hipfftMakePlanMany(plan, 1, n,
                                  nullptr, 1, 0,
                                  nullptr, 1, 0,
                                  HIPFFT_C2C, 1, &workSize));
    size_t queriedSize = 0;
    CUFFT_CHECK(hipfftGetSize(plan, &queriedSize));
    ASSERT_EQ(workSize, queriedSize);
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, SetWorkArea) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    void* workArea;
    CUDA_CHECK(hipMalloc(&workArea, 1024));
    CUFFT_CHECK(hipfftSetWorkArea(plan, workArea));
    CUFFT_CHECK(hipfftDestroy(plan));
    CUDA_CHECK(hipFree(workArea));
}

#if CUDART_VERSION <= 9000
TEST(cuFFT, SetCompatibilityMode) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    CUFFT_CHECK(cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));
    CUFFT_CHECK(hipfftDestroy(plan));
}
#endif

TEST(cuFFT, SetAutoAllocation) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    CUFFT_CHECK(hipfftSetAutoAllocation(plan, 0));  // Manual mode
    CUFFT_CHECK(hipfftSetAutoAllocation(plan, 1));  // Back to default
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, GetVersion) {
    int version = 0;
    CUFFT_CHECK(hipfftGetVersion(&version));
    ASSERT_GT(version, 0);
}

TEST(cuFFT, SetStream) {
    hipfftHandle plan;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    CUFFT_CHECK(hipfftPlan1d(&plan, 16, HIPFFT_C2C, 1));
    CUFFT_CHECK(hipfftSetStream(plan, stream));

    CUDA_CHECK(hipStreamDestroy(stream));
    CUFFT_CHECK(hipfftDestroy(plan));
}

#if __CUDA_API_VERSION >= 7000
TEST(cuFFT, GetProperty) {
    size_t property = 0;
    CUFFT_CHECK(hipfftGetProperty(CUFFT_MAJOR_VERSION, &property));
    ASSERT_GT(property, 0);
}
#endif

TEST(cuFFT, ExecZ2Z) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan1d(&plan, 8, HIPFFT_Z2Z, 1));

    hipfftDoubleComplex* data;
    CUDA_CHECK(hipMalloc(&data, sizeof(hipfftDoubleComplex) * 8));

    hipfftDoubleComplex h_data[8];
    for (int i = 0; i < 8; i++) {
        h_data[i].x = i;
        h_data[i].y = 0;
    }
    CUDA_CHECK(hipMemcpy(data, h_data, sizeof(h_data), hipMemcpyHostToDevice));

    CUFFT_CHECK(hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD));

    CUDA_CHECK(hipMemcpy(h_data, data, sizeof(h_data), hipMemcpyDeviceToHost));

    CUFFT_CHECK(hipfftDestroy(plan));
    CUDA_CHECK(hipFree(data));
}

TEST(cuFFT, ExecD2Z) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan1d(&plan, 8, HIPFFT_D2Z, 1));

    double* idata;
    hipfftDoubleComplex* odata;
    CUDA_CHECK(hipMalloc(&idata, sizeof(double) * 8));
    CUDA_CHECK(hipMalloc(&odata, sizeof(hipfftDoubleComplex) * 5));

    double h_idata[8];
    for (int i = 0; i < 8; i++) h_idata[i] = i;
    CUDA_CHECK(hipMemcpy(idata, h_idata, sizeof(h_idata), hipMemcpyHostToDevice));

    CUFFT_CHECK(hipfftExecD2Z(plan, idata, odata));

    // Optionally copy output back to host for inspection
    hipfftDoubleComplex h_odata[5];
    CUDA_CHECK(hipMemcpy(h_odata, odata, sizeof(h_odata), hipMemcpyDeviceToHost));

    CUFFT_CHECK(hipfftDestroy(plan));
    CUDA_CHECK(hipFree(idata));
    CUDA_CHECK(hipFree(odata));
}

TEST(cuFFT, ExecZ2D) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan1d(&plan, 8, HIPFFT_Z2D, 1));

    hipfftDoubleComplex* idata;
    double* odata;
    CUDA_CHECK(hipMalloc(&idata, sizeof(hipfftDoubleComplex) * 5));
    CUDA_CHECK(hipMalloc(&odata, sizeof(double) * 8));

    hipfftDoubleComplex h_idata[5];
    for (int i = 0; i < 5; i++) {
        h_idata[i].x = i;
        h_idata[i].y = 0;
    }
    CUDA_CHECK(hipMemcpy(idata, h_idata, sizeof(h_idata), hipMemcpyHostToDevice));

    CUFFT_CHECK(hipfftExecZ2D(plan, idata, odata));

    double h_odata[8];
    CUDA_CHECK(hipMemcpy(h_odata, odata, sizeof(h_odata), hipMemcpyDeviceToHost));

    CUFFT_CHECK(hipfftDestroy(plan));
    CUDA_CHECK(hipFree(idata));
    CUDA_CHECK(hipFree(odata));
}

TEST(cuFFT, XtSetGPUs) {
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount < 2)
        GTEST_SKIP() << "Test requires multiple GPUs";

    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));  // Step 1: Create

    int gpus[1] = {0};
    CUFFT_CHECK(hipfftXtSetGPUs(plan, 1, gpus));  // Step 2: Set GPUs

    int n[1] = {8};  // 1D FFT of length 8
    size_t workSize = 0;
    CUFFT_CHECK(hipfftMakePlanMany(plan, 1, n,
                                  nullptr, 1, 0,
                                  nullptr, 1, 0,
                                  HIPFFT_C2C, 1, &workSize));  // Step 3: Make Plan

    CUFFT_CHECK(hipfftDestroy(plan));
}

// Not supported by GVirtuS
TEST(cuFFT, XtMakePlanMany) {
    hipfftHandle plan;
    CUFFT_CHECK(hipfftCreate(&plan));
    long long int n[] = {8};
    long long int inembed[] = {8};
    long long int onembed[] = {8};
    long long int howmany = 1;
    size_t workSize = 0;
    // This should return HIPFFT_NOT_IMPLEMENTED
    // because hipfftXtMakePlanMany is not supported in GVirtuS
    ASSERT_EQ(hipfftXtMakePlanMany(plan, 1, n,
                                    inembed, 1, 8, HIP_C_32F,
                                    onembed, 1, 8, HIP_C_32F,
                                    howmany, &workSize, HIP_C_32F), HIPFFT_NOT_IMPLEMENTED);
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, XtMallocFree) {
    constexpr size_t N = 8;
    hipfftHandle plan;
    hipLibXtDesc* deviceDesc = nullptr;

    // Create 1D plan
    CUFFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

    // Allocate device memory using XtMalloc
    CUFFT_CHECK(hipfftXtMalloc(plan, &deviceDesc, HIPFFT_XT_FORMAT_INPLACE));
    ASSERT_NE(deviceDesc, nullptr);

    // Free the memory using XtFree
    CUFFT_CHECK(hipfftXtFree(deviceDesc));

    // Destroy plan
    CUFFT_CHECK(hipfftDestroy(plan));
}

TEST(cuFFT, CufftXtMallocMemcpyFree) {
    // int deviceCount = 0;
    // CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    // if (deviceCount < 2)
    //     GTEST_SKIP() << "Test requires multiple GPUs";

    constexpr size_t N = 8;
    hipfftHandle plan;
    hipLibXtDesc* deviceDesc = nullptr;
    float* hostData = new float[N];

    for (size_t i = 0; i < N; ++i)
        hostData[i] = static_cast<float>(i);

    // Create cuFFT plan
    CUFFT_CHECK(hipfftPlan1d(&plan, N, HIPFFT_R2C, 1));

    // Allocate memory using Xt API
    CUFFT_CHECK(hipfftXtMalloc(plan, &deviceDesc, HIPFFT_XT_FORMAT_INPLACE));

    // Use the opaque descriptor directly in XtMemcpy
    CUFFT_CHECK(hipfftXtMemcpy(plan,
                              reinterpret_cast<void*>(deviceDesc),
                              reinterpret_cast<void*>(hostData),
                              HIPFFT_COPY_HOST_TO_DEVICE));

    // Clear hostData before copying back
    for (size_t i = 0; i < N; ++i)
        hostData[i] = 0.0f;

    // Copy back from device to host
    CUFFT_CHECK(hipfftXtMemcpy(plan,
                              reinterpret_cast<void*>(hostData),
                              reinterpret_cast<void*>(deviceDesc),
                              HIPFFT_COPY_DEVICE_TO_HOST));

    // Check round-trip correctness
    for (size_t i = 0; i < N; ++i)
        ASSERT_FLOAT_EQ(hostData[i], static_cast<float>(i));

    // Cleanup
    // CUFFT_CHECK(hipfftXtFree(deviceDesc));
    CUFFT_CHECK(hipfftDestroy(plan));
    delete[] hostData;
}

TEST(cuFFT, CufftXtExecDescriptorC2C) {
    // int deviceCount = 0;
    // CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    // if (deviceCount < 2)
    //     GTEST_SKIP() << "Test requires multiple GPUs";

    constexpr size_t N = 8;
    hipfftHandle planFwd, planInv;
    hipLibXtDesc* deviceDescFwd = nullptr;
    hipLibXtDesc* deviceDescInv = nullptr;
    hipfftComplex* hostData = new hipfftComplex[N];

    // Initialize host input with complex numbers
    for (size_t i = 0; i < N; ++i) {
        hostData[i].x = static_cast<float>(i);
        hostData[i].y = 0.0f;
    }

    // Create forward and inverse cuFFT plans
    CUFFT_CHECK(hipfftPlan1d(&planFwd, N, HIPFFT_C2C, 1));
    CUFFT_CHECK(hipfftPlan1d(&planInv, N, HIPFFT_C2C, 1));

    // Allocate Xt memory
    CUFFT_CHECK(hipfftXtMalloc(planFwd, &deviceDescFwd, HIPFFT_XT_FORMAT_INPLACE));
    CUFFT_CHECK(hipfftXtMalloc(planInv, &deviceDescInv, HIPFFT_XT_FORMAT_INPLACE));

    // Host to device
    CUFFT_CHECK(hipfftXtMemcpy(planFwd,
                              reinterpret_cast<void*>(deviceDescFwd),
                              reinterpret_cast<void*>(hostData),
                              HIPFFT_COPY_HOST_TO_DEVICE));

    // Execute forward FFT
    CUFFT_CHECK(hipfftXtExecDescriptorC2C(planFwd, deviceDescFwd, deviceDescInv, HIPFFT_FORWARD));

    // Execute inverse FFT (in-place)
    CUFFT_CHECK(hipfftXtExecDescriptorC2C(planInv, deviceDescInv, deviceDescFwd, HIPFFT_BACKWARD));

    // Copy result back to host
    CUFFT_CHECK(hipfftXtMemcpy(planInv,
                              reinterpret_cast<void*>(hostData),
                              reinterpret_cast<void*>(deviceDescFwd),
                              HIPFFT_COPY_DEVICE_TO_HOST));

    // Check that output matches original input (within tolerance)
    for (size_t i = 0; i < N; ++i) {
        ASSERT_NEAR(hostData[i].x / N, static_cast<float>(i), 1e-3);
        ASSERT_NEAR(hostData[i].y / N, 0.0f, 1e-3);
    }

    // Cleanup
    CUFFT_CHECK(hipfftXtFree(deviceDescFwd));
    CUFFT_CHECK(hipfftXtFree(deviceDescInv));
    CUFFT_CHECK(hipfftDestroy(planFwd));
    CUFFT_CHECK(hipfftDestroy(planInv));
    delete[] hostData;
}

