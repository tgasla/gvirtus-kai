#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)
#define CUBLAS_CHECK(err) ASSERT_EQ((err), HIPBLAS_STATUS_SUCCESS)

TEST(cuBLAS, CreateDestroy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, SetStreamDestroy) {
    hipblasHandle_t handle;
    hipStream_t stream;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUDA_CHECK(hipStreamCreate(&stream));
    CUBLAS_CHECK(hipblasSetStream(handle, stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Sgemm) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Matrix size 2x2 for simplicity
    const int N = 2;
    float h_A[] = {1, 2, 3, 4};  // col-major 2x2
    float h_B[] = {5, 6, 7, 8};
    float h_C[4] = {0};

    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, N * N * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_C, 0, N * N * sizeof(float)));

    float alpha = 1.0f, beta = 0.0f;

    // SGEMM: C = alpha * A * B + beta * C
    CUBLAS_CHECK(hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha,
                             d_A, N,
                             d_B, N,
                             &beta,
                             d_C, N));

    CUDA_CHECK(hipMemcpy(h_C, d_C, sizeof(h_C), hipMemcpyDeviceToHost));

    // Check a few expected values
    ASSERT_FLOAT_EQ(h_C[0], 19.0f); // 1*5+3*6=5+18=23 (wait: col-major, careful!)
    // Let's calculate correct expected values for col-major:
    // C = A * B, with A and B col-major:
    // A = |1 3|
    //     |2 4|
    // B = |5 7|
    //     |6 8|
    // C[0,0] = 1*5 + 3*6 = 5 + 18 = 23
    // C[1,0] = 2*5 + 4*6 = 10 + 24 = 34
    // C[0,1] = 1*7 + 3*8 = 7 + 24 = 31
    // C[1,1] = 2*7 + 4*8 = 14 + 32 = 46

    ASSERT_FLOAT_EQ(h_C[0], 23.0f);
    ASSERT_FLOAT_EQ(h_C[1], 34.0f);
    ASSERT_FLOAT_EQ(h_C[2], 31.0f);
    ASSERT_FLOAT_EQ(h_C[3], 46.0f);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Sgemv) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    const int M = 2, N = 2;
    float h_A[] = {1, 3, 2, 4};  // 2x2 col-major
    float h_x[] = {1, 2};
    float h_y[] = {0, 0};

    float *d_A, *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_A, M * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_x, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, M * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_y, 0, M * sizeof(float)));

    float alpha = 1.0f, beta = 0.0f;

    CUBLAS_CHECK(hipblasSgemv(handle,
                             HIPBLAS_OP_N,
                             M, N,
                             &alpha,
                             d_A, M,
                             d_x, 1,
                             &beta,
                             d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, M * sizeof(float), hipMemcpyDeviceToHost));

    // y = A*x = [1*1+3*2, 2*1+4*2] = [1+6, 2+8] = [7, 10]
    ASSERT_FLOAT_EQ(h_y[0], 7.0f);
    ASSERT_FLOAT_EQ(h_y[1], 10.0f);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Saxpy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {1, 2, 3};
    float h_y[] = {4, 5, 6};

    float *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    float alpha = 2.0f;

    CUBLAS_CHECK(hipblasSaxpy(handle, n, &alpha, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    // y = y + alpha*x = [4+2*1,5+2*2,6+2*3] = [6,9,12]
    ASSERT_FLOAT_EQ(h_y[0], 6.0f);
    ASSERT_FLOAT_EQ(h_y[1], 9.0f);
    ASSERT_FLOAT_EQ(h_y[2], 12.0f);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Scopy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {1, 2, 3};
    float h_y[] = {0, 0, 0};

    float *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasScopy(handle, n, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; ++i) {
        ASSERT_FLOAT_EQ(h_y[i], h_x[i]);
    }

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Snrm2) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {3, 4, 0};

    float *d_x;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));

    float result = 0;
    CUBLAS_CHECK(hipblasSnrm2(handle, n, d_x, 1, &result));

    ASSERT_NEAR(result, 5.0f, 1e-5);

    CUDA_CHECK(hipFree(d_x));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Sdot) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    float h_x[] = {1, 2, 3};
    float h_y[] = {4, 5, 6};

    float *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    float result = 0;
    CUBLAS_CHECK(hipblasSdot(handle, n, d_x, 1, d_y, 1, &result));

    // dot product = 1*4 + 2*5 + 3*6 = 32
    ASSERT_FLOAT_EQ(result, 32.0f);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dgemm) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int m = 2, n = 3, k = 2;
    double alpha = 1.0, beta = 0.0;

    // A is m x k
    double h_A[] = {1.0, 2.0,
                    3.0, 4.0};

    // B is k x n
    double h_B[] = {5.0, 6.0, 7.0,
                    8.0, 9.0, 10.0};

    // C is m x n
    double h_C[6] = {0};

    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, m * k * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B, k * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, m * n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeof(h_B), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, h_C, sizeof(h_C), hipMemcpyHostToDevice));

    // Perform C = alpha * A * B + beta * C
    // Note: cuBLAS is column-major by default, so we can either transpose matrices or switch order:
    // Using row-major layout, specify operation flags as HIPBLAS_OP_T to transpose inputs

    // Here, to keep it simple, we use the matrices as column-major:
    CUBLAS_CHECK(hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             m, n, k,
                             &alpha,
                             d_A, m,
                             d_B, k,
                             &beta,
                             d_C, m));

    CUDA_CHECK(hipMemcpy(h_C, d_C, sizeof(h_C), hipMemcpyDeviceToHost));

    // Expected result:
    // C = [1*5+2*8, 1*6+2*9, 1*7+2*10
    //      3*5+4*8, 3*6+4*9, 3*7+4*10]
    // = [21, 24, 27
    //    47, 54, 61]

    double expected[] = {21, 24, 27, 47, 54, 61};
    for (int i = 0; i < m * n; ++i) {
        ASSERT_NEAR(h_C[i], expected[i], 1e-9);
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dgemv) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int m = 2, n = 3;
    double alpha = 1.0, beta = 0.0;

    // A is m x n
    double h_A[] = {1.0, 2.0, 3.0,
                    4.0, 5.0, 6.0};

    double h_x[] = {1.0, 1.0, 1.0};
    double h_y[] = {0.0, 0.0};

    double *d_A, *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_A, m * n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, m * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeof(h_A), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    // y = alpha * A * x + beta * y
    CUBLAS_CHECK(hipblasDgemv(handle,
                             HIPBLAS_OP_N,
                             m, n,
                             &alpha,
                             d_A, m,
                             d_x, 1,
                             &beta,
                             d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    // Expected result:
    // y[0] = 1*1 + 2*1 + 3*1 = 6
    // y[1] = 4*1 + 5*1 + 6*1 = 15

    ASSERT_NEAR(h_y[0], 6.0, 1e-9);
    ASSERT_NEAR(h_y[1], 15.0, 1e-9);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(CuBLAS, Daxpy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double alpha = 2.0;
    double h_x[] = {1.0, 2.0, 3.0};
    double h_y[] = {4.0, 5.0, 6.0};

    double *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    // y = alpha * x + y
    CUBLAS_CHECK(hipblasDaxpy(handle, n, &alpha, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    // Expected y = [4+2*1, 5+2*2, 6+2*3] = [6, 9, 12]
    ASSERT_NEAR(h_y[0], 6.0, 1e-9);
    ASSERT_NEAR(h_y[1], 9.0, 1e-9);
    ASSERT_NEAR(h_y[2], 12.0, 1e-9);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dcopy) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double h_x[] = {1.0, 2.0, 3.0};
    double h_y[] = {0.0, 0.0, 0.0};

    double *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasDcopy(handle, n, d_x, 1, d_y, 1));

    CUDA_CHECK(hipMemcpy(h_y, d_y, sizeof(h_y), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; ++i) {
        ASSERT_DOUBLE_EQ(h_y[i], h_x[i]);
    }

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Dnrm2) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double h_x[] = {3.0, 4.0, 0.0};

    double *d_x;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));

    double result = 0;
    CUBLAS_CHECK(hipblasDnrm2(handle, n, d_x, 1, &result));

    ASSERT_NEAR(result, 5.0, 1e-9);

    CUDA_CHECK(hipFree(d_x));
    CUBLAS_CHECK(hipblasDestroy(handle));
}

TEST(cuBLAS, Ddot) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    int n = 3;
    double h_x[] = {1.0, 2.0, 3.0};
    double h_y[] = {4.0, 5.0, 6.0};

    double *d_x, *d_y;
    CUDA_CHECK(hipMalloc(&d_x, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_y, n * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_x, h_x, sizeof(h_x), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y, h_y, sizeof(h_y), hipMemcpyHostToDevice));

    double result = 0;
    CUBLAS_CHECK(hipblasDdot(handle, n, d_x, 1, d_y, 1, &result));

    // dot product = 1*4 + 2*5 + 3*6 = 32
    ASSERT_DOUBLE_EQ(result, 32.0);

    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUBLAS_CHECK(hipblasDestroy(handle));
}
