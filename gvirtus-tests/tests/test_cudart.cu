#include <gtest/gtest.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err) ASSERT_EQ((err), hipSuccess)

TEST(cudaRT, MallocFree) {
    void* devPtr = nullptr;
    CUDA_CHECK(hipMalloc(&devPtr, 1024));
    CUDA_CHECK(hipFree(devPtr));
}

TEST(cudaRT, MemcpySync) {
    int h_src = 42;
    int h_dst = 0;
    int* d_ptr;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_ptr, &h_src, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(&h_dst, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(h_dst, 42);

    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, MemcpyAsync) {
    int h_src = 24;
    int h_dst = 0;
    int* d_ptr;
    hipStream_t stream;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));
    CUDA_CHECK(hipStreamCreate(&stream));

    CUDA_CHECK(hipMemcpyAsync(d_ptr, &h_src, sizeof(int), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(&h_dst, d_ptr, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));
    EXPECT_EQ(h_dst, 24);

    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, Memset) {
    int* d_ptr;
    CUDA_CHECK(hipMalloc(&d_ptr, sizeof(int)));
    CUDA_CHECK(hipMemset(d_ptr, 0, sizeof(int)));

    int h_val = 1;
    CUDA_CHECK(hipMemcpy(&h_val, d_ptr, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(h_val, 0);

    CUDA_CHECK(hipFree(d_ptr));
}

TEST(cudaRT, StreamCreateDestroySynchronize) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

TEST(cudaRT, GetDevice) {
    int device;
    CUDA_CHECK(hipGetDevice(&device));
}

TEST(cudaRT, SetDevice) {
    int device = 0;
    CUDA_CHECK(hipSetDevice(device));
}

TEST(cudaRT, DeviceSynchronize) {
    CUDA_CHECK(hipDeviceSynchronize());
}

// TEST(cudaRT, EventCreateRecordSynchronizeElapsedTime) {
//     hipEvent_t start, stop;
//     CUDA_CHECK(hipEventCreate(&start));
//     CUDA_CHECK(hipEventCreate(&stop));

//     CUDA_CHECK(hipEventRecord(start));
//     CUDA_CHECK(hipEventRecord(stop));

//     CUDA_CHECK(hipEventSynchronize(stop));

//     float elapsed_ms = 0;
//     CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
//     EXPECT_GE(elapsed_ms, 0.0f);

//     CUDA_CHECK(hipEventDestroy(start));
//     CUDA_CHECK(hipEventDestroy(stop));
// }
