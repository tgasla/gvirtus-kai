#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <iostream>
#include <hiprand.h>

TEST(CurandTest, CreateDestroyGenerator) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
}

TEST(CurandTest, CreateDestroyGeneratorHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
}

TEST(CurandTest, SetSeed) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
}

TEST(CurandTest, GenerateDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 10;
    unsigned int* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(unsigned int)), hipSuccess);

    ASSERT_EQ(hiprandGenerate(generator, output, n), HIPRAND_STATUS_SUCCESS);

    unsigned int host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(unsigned int), hipMemcpyDeviceToHost), hipSuccess);

    bool all_zero = true;
    for (size_t i = 0; i < n; ++i) {
        if (host_output[i] != 0) {
            all_zero = false;
            break;
        }
    }
    EXPECT_FALSE(all_zero); // Generated numbers should not all be zero

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GenerateHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 10;
    unsigned int* output = (unsigned int*)malloc(n * sizeof(unsigned int));
    ASSERT_NE(output, nullptr);

    ASSERT_EQ(hiprandGenerate(generator, output, n), HIPRAND_STATUS_SUCCESS);

    bool all_zero = true;
    for (size_t i = 0; i < n; ++i) {
        if (output[i] != 0) {
            all_zero = false;
            break;
        }
    }
    EXPECT_FALSE(all_zero); // Generated numbers should not all be zero

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}

TEST(CurandTest, GenerateLongLongDevice) {
    hiprandGenerator_t generator;
    const size_t num = 10;

    // Create a quasi-random number generator
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SOBOL64), HIPRAND_STATUS_SUCCESS) << "Failed to create generator";

    // Set dimensions (required for quasi generators)
    ASSERT_EQ(hiprandSetQuasiRandomGeneratorDimensions(generator, 1), HIPRAND_STATUS_SUCCESS) << "Failed to set dimensions";

    // Allocate device memory
    unsigned long long* d_output = nullptr;
    ASSERT_EQ(hipMalloc(&d_output, num * sizeof(unsigned long long)), hipSuccess) << "Failed to allocate device memory";

    // Generate quasi-random numbers
    ASSERT_EQ(hiprandGenerateLongLong(generator, d_output, num), HIPRAND_STATUS_SUCCESS) << "hiprandGenerateLongLong failed";

    // Copy results back to host for checking
    unsigned long long h_output[num];
    ASSERT_EQ(hipMemcpy(h_output, d_output, num * sizeof(unsigned long long), hipMemcpyDeviceToHost), hipSuccess) << "Failed to copy data from device to host";

    std::cout << "Generated long long values (quasi): ";
    for (size_t i = 0; i < num; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    ASSERT_EQ(hipFree(d_output), hipSuccess) << "Failed to free device memory";
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS) << "Failed to destroy generator";
}

TEST(CurandTest, GenerateLongLongHost) {
    hiprandGenerator_t generator;
    const size_t num = 10;

    // Create a QUASI-random number generator (host generator)
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_QUASI_SOBOL64), HIPRAND_STATUS_SUCCESS) << "Failed to create QUASI generator";

     // Set dimensions (required for quasi generators)
    ASSERT_EQ(hiprandSetQuasiRandomGeneratorDimensions(generator, 1), HIPRAND_STATUS_SUCCESS) << "Failed to set dimensions";

    // Allocate host memory for output
    unsigned long long* h_output = new unsigned long long[num];

    // Generate random numbers on host
    ASSERT_EQ(hiprandGenerateLongLong(generator, h_output, num), HIPRAND_STATUS_SUCCESS) << "hiprandGenerateLongLong failed";

    std::cout << "Generated long long values (host, QUASI): ";
    for (size_t i = 0; i < num; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_output;
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS) << "Failed to destroy generator";
}

TEST(CurandTest, GenerateUniformDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 10;
    float* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(float)), hipSuccess);

    ASSERT_EQ(hiprandGenerateUniform(generator, output, n), HIPRAND_STATUS_SUCCESS);

    float host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(float), hipMemcpyDeviceToHost), hipSuccess);

    for (size_t i = 0; i < n; ++i) {
        EXPECT_GE(host_output[i], 0.0f);
        EXPECT_LT(host_output[i], 1.0f);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GenerateUniformHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 10;
    float* output = (float*)malloc(n * sizeof(float));
    ASSERT_NE(output, nullptr);

    ASSERT_EQ(hiprandGenerateUniform(generator, output, n), HIPRAND_STATUS_SUCCESS);

    for (size_t i = 0; i < n; ++i) {
        EXPECT_GE(output[i], 0.0f);
        EXPECT_LT(output[i], 1.0f);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}

TEST(CurandTest, GenerateNormalDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 9012ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;  // Larger sample for stats
    float* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(float)), hipSuccess);

    const float mean = 5.0f;
    const float stddev = 2.0f;
    ASSERT_EQ(hiprandGenerateNormal(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    float host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(float), hipMemcpyDeviceToHost), hipSuccess);

    // Basic sanity: check mean and stddev roughly
    float sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        sum += host_output[i];
    }
    float sample_mean = sum / n;
    EXPECT_NEAR(sample_mean, mean, 0.2f);

    float variance_sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        float diff = host_output[i] - mean;
        variance_sum += diff * diff;
    }
    float sample_stddev = sqrt(variance_sum / n);
    EXPECT_NEAR(sample_stddev, stddev, 0.3f);

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GenerateNormalHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 9012ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    float* output = (float*)malloc(n * sizeof(float));
    ASSERT_NE(output, nullptr);

    const float mean = 5.0f;
    const float stddev = 2.0f;
    ASSERT_EQ(hiprandGenerateNormal(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    // Basic sanity: check mean and stddev roughly
    float sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        sum += output[i];
    }
    float sample_mean = sum / n;
    EXPECT_NEAR(sample_mean, mean, 0.2f);

    float variance_sum = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        float diff = output[i] - mean;
        variance_sum += diff * diff;
    }
    float sample_stddev = sqrt(variance_sum / n);
    EXPECT_NEAR(sample_stddev, stddev, 0.3f);

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}

TEST(CurandTest, GenerateLogNormalDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 8642ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    float* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(float)), hipSuccess);

    const float mean = 0.0f;
    const float stddev = 0.5f;
    ASSERT_EQ(hiprandGenerateLogNormal(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    float host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(float), hipMemcpyDeviceToHost), hipSuccess);

    // All values should be positive
    for (size_t i = 0; i < n; ++i) {
        EXPECT_GT(host_output[i], 0.0f);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GenerateLogNormalHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 8642ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    float* output = (float*)malloc(n * sizeof(float));
    ASSERT_NE(output, nullptr);

    const float mean = 0.0f;
    const float stddev = 0.5f;
    ASSERT_EQ(hiprandGenerateLogNormal(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    // All values should be positive
    for (size_t i = 0; i < n; ++i) {
        EXPECT_GT(output[i], 0.0f);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}

TEST(CurandTest, GeneratePoissonDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 3456ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    unsigned int* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(unsigned int)), hipSuccess);

    double lambda = 4.5;
    ASSERT_EQ(hiprandGeneratePoisson(generator, output, n, lambda), HIPRAND_STATUS_SUCCESS);

    unsigned int host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(unsigned int), hipMemcpyDeviceToHost), hipSuccess);

    // Basic sanity checks: values >= 0 and mean close to lambda
    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        EXPECT_GE(host_output[i], 0u);
        sum += host_output[i];
    }
    double sample_mean = sum / n;
    EXPECT_NEAR(sample_mean, lambda, 0.2);

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GeneratePoissonHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 3456ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    unsigned int* output = (unsigned int*)malloc(n * sizeof(unsigned int));
    ASSERT_NE(output, nullptr);

    double lambda = 4.5;
    ASSERT_EQ(hiprandGeneratePoisson(generator, output, n, lambda), HIPRAND_STATUS_SUCCESS);

    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        EXPECT_GE(output[i], 0u);
        sum += output[i];
    }
    double sample_mean = sum / n;
    EXPECT_NEAR(sample_mean, lambda, 0.2);

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}

TEST(CurandTest, GenerateUniformDoubleDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 7890ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 10;
    double* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(double)), hipSuccess);

    ASSERT_EQ(hiprandGenerateUniformDouble(generator, output, n), HIPRAND_STATUS_SUCCESS);

    double host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(double), hipMemcpyDeviceToHost), hipSuccess);

    for (size_t i = 0; i < n; ++i) {
        EXPECT_GE(host_output[i], 0.0);
        EXPECT_LT(host_output[i], 1.0);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GenerateUniformDoubleHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 7890ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 10;
    double* output = (double*)malloc(n * sizeof(double));
    ASSERT_NE(output, nullptr);

    ASSERT_EQ(hiprandGenerateUniformDouble(generator, output, n), HIPRAND_STATUS_SUCCESS);

    for (size_t i = 0; i < n; ++i) {
        EXPECT_GE(output[i], 0.0);
        EXPECT_LT(output[i], 1.0);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}

TEST(CurandTest, GenerateNormalDoubleDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 2468ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    double* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(double)), hipSuccess);

    const double mean = 10.0;
    const double stddev = 3.0;
    ASSERT_EQ(hiprandGenerateNormalDouble(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    double host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(double), hipMemcpyDeviceToHost), hipSuccess);

    // Basic sanity: check mean and stddev roughly
    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        sum += host_output[i];
    }
    double sample_mean = sum / n;
    EXPECT_NEAR(sample_mean, mean, 0.2);

    double variance_sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        double diff = host_output[i] - mean;
        variance_sum += diff * diff;
    }
    double sample_stddev = sqrt(variance_sum / n);
    EXPECT_NEAR(sample_stddev, stddev, 0.3);

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GenerateNormalDoubleHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 2468ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    double* output = (double*)malloc(n * sizeof(double));
    ASSERT_NE(output, nullptr);

    const double mean = 10.0;
    const double stddev = 3.0;
    ASSERT_EQ(hiprandGenerateNormalDouble(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    // Basic sanity: check mean and stddev roughly
    double sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        sum += output[i];
    }
    double sample_mean = sum / n;
    EXPECT_NEAR(sample_mean, mean, 0.2);

    double variance_sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        double diff = output[i] - mean;
        variance_sum += diff * diff;
    }
    double sample_stddev = sqrt(variance_sum / n);
    EXPECT_NEAR(sample_stddev, stddev, 0.3);

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}

TEST(CurandTest, GenerateLogNormalDoubleDevice) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1357ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    double* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(double)), hipSuccess);

    const double mean = 0.0;   // mean of underlying normal
    const double stddev = 0.5; // stddev of underlying normal
    ASSERT_EQ(hiprandGenerateLogNormalDouble(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    double host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(double), hipMemcpyDeviceToHost), hipSuccess);

    // All outputs should be positive
    for (size_t i = 0; i < n; ++i) {
        EXPECT_GT(host_output[i], 0.0);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

TEST(CurandTest, GenerateLogNormalDoubleHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1357ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 1000;
    double* output = (double*)malloc(n * sizeof(double));
    ASSERT_NE(output, nullptr);

    const double mean = 0.0;
    const double stddev = 0.5;
    ASSERT_EQ(hiprandGenerateLogNormalDouble(generator, output, n, mean, stddev), HIPRAND_STATUS_SUCCESS);

    // All outputs should be positive
    for (size_t i = 0; i < n; ++i) {
        EXPECT_GT(output[i], 0.0);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    free(output);
}