#include <gtest/gtest.h>
#include <iostream>
#include <hiprand.h>

TEST(CurandTest, CreateGenerator) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
}

TEST(CurandTest, CreateGeneratorHost) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGeneratorHost(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
}

TEST(CurandTest, SetSeed) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
}

TEST(CurandTest, GenerateUniform) {
    hiprandGenerator_t generator;
    ASSERT_EQ(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL), HIPRAND_STATUS_SUCCESS);

    const size_t n = 10;
    float* output;
    ASSERT_EQ(hipMalloc(&output, n * sizeof(float)), hipSuccess);

    ASSERT_EQ(hiprandGenerateUniform(generator, output, n), HIPRAND_STATUS_SUCCESS);

    float host_output[n];
    ASSERT_EQ(hipMemcpy(host_output, output, n * sizeof(float), hipMemcpyDeviceToHost), hipSuccess);

    for (size_t i = 0; i < n; ++i) {
        EXPECT_GE(host_output[i], 0.0f);
        EXPECT_LT(host_output[i], 1.0f);
    }

    ASSERT_EQ(hiprandDestroyGenerator(generator), HIPRAND_STATUS_SUCCESS);
    ASSERT_EQ(hipFree(output), hipSuccess);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}