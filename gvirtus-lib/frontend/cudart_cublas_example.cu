#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main() {
    const int size = 5;
    float a[size] = {1, 2, 3, 4, 5};
    float b[size] = {6, 7, 8, 9, 10};
    float c[size]; // To store the result

    float *d_a, *d_b;

    // Allocate memory on device
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));

    // Copy data to device
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform c = a + b using cublasSaxpy (b = 1.0 * a + b)
    float alpha = 1.0f;
    hipblasSaxpy(handle, size, &alpha, d_a, 1, d_b, 1);

    // Copy result to host
    hipMemcpy(c, d_b, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < size; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
