
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand_kernel.h>
#include <cudnn.h>

int main() {
    // ======== CUDA Runtime ========
    float *d_data;
    hipMalloc(&d_data, 10 * sizeof(float));
    hipMemset(d_data, 0, 10 * sizeof(float));
    std::cout << "cudart: Allocated and zeroed memory.\n";

    // ======== cuBLAS ========
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    float alpha = 1.0f;
    float *x, *y;
    hipMalloc(&x, 5 * sizeof(float));
    hipMalloc(&y, 5 * sizeof(float));
    hipblasSaxpy(cublasHandle, 5, &alpha, x, 1, y, 1);
    std::cout << "cuBLAS: Performed SAXPY.\n";
    hipblasDestroy(cublasHandle);

    // ======== cuFFT ========
    hipfftHandle fftPlan;
    hipfftComplex *fftData;
    hipMalloc(&fftData, 8 * sizeof(hipfftComplex));
    hipfftPlan1d(&fftPlan, 8, HIPFFT_C2C, 1);
    hipfftExecC2C(fftPlan, fftData, fftData, HIPFFT_FORWARD);
    std::cout << "cuFFT: Executed 1D FFT.\n";
    hipfftDestroy(fftPlan);

    // ======== cuRAND ========
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateUniform(gen, d_data, 10);
    std::cout << "cuRAND: Generated uniform random numbers.\n";
    //curandDestroyGenerator(gen);

    // ======== cuDNN ========
    //cudnnHandle_t cudnnHandle;
    //cudnnCreate(&cudnnHandle);
    //std::cout << "created handle\n";
    //cudnnTensorDescriptor_t desc;
    //cudnnCreateTensorDescriptor(&desc);
    //std::cout << "created tensor descriptor\n";
    //cudnnSetTensor4dDescriptor(desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, 1, 1, 1);
    //std::cout << "cuDNN: Created tensor descriptor.\n";
    //cudnnDestroyTensorDescriptor(desc);
    //cudnnDestroy(cudnnHandle);


    // ======== Cleanup ========
    //cudaFree(d_data);
    //cudaFree(x);
    //cudaFree(y);
    //cudaFree(fftData);

    return 0;
}
