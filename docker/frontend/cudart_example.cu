#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to add two vectors
__global__ void addVectors(int *a, int *b, int *c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int size = 5;
    int a[size] = {1, 2, 3, 4, 5};
    int b[size] = {6, 7, 8, 9, 10};
    int c[size]; // To store the result

    int *d_a, *d_b, *d_c; // Device pointers

    // Allocate memory on device
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 1 block of size 'size' threads
    addVectors<<<1, size>>>(d_a, d_b, d_c, size);

    // Copy result back from device to host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < size; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
